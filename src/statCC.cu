#define ARMA_ALLOW_FAKE_GCC

#define VIENNACL_WITH_CUDA
#define VIENNACL_WITH_OPENMP
#define VIENNACL_WITH_ARMADILLO 1

#include <RcppArmadillo.h>
// #include <RcppArmadilloExtensions/sample.h>
using namespace Rcpp;

// ViennaCL headers
#include "viennacl/vector.hpp"
#include "viennacl/matrix.hpp"
#include "viennacl/forwards.h"
#include "viennacl/matrix_proxy.hpp"
#include "viennacl/linalg/inner_prod.hpp"
#include "viennacl/linalg/sum.hpp"

#include "hsic.h"


// [[Rcpp::plugins(cpp11)]]
// [[Rcpp::depends(RcppArmadillo)]]
double statCC(arma::vec sample, arma::mat replicates, arma::field<arma::mat> K){

    // Compute the sum kernel
    int n = sample.size();
    arma::vec stat(replicates.n_cols);
    arma::mat Ksum(n, n, arma::fill::zeros);
    for (int r = 0; r < K.n_elem; ++r){
       Ksum += K(r);
    }
    Ksum = quadHSIC(Ksum);


    // CUDA section
    double* hsicCUDA, replicatesCUDA, prodCUDA, sampleCUDA;

    // Allocate all our host-side (CPU) and device-side (GPU) data
    hipMallocManaged( (void **)&hsicCUDA, n * n * sizeof( double ));
    hipMallocManaged( (void **)&replicatesCUDA, replicates.n_rows * replicates.n_cols * sizeof( double ));
    hipMallocManaged( (void **)&prodCUDA, replicates.n_rows * replicates.n_cols * sizeof( double ));
    hipMallocManaged( (void **)&sampleCUDA, n * sizeof( double ));

    // Copy data to CUDA objects
    hipMemcpy(hsicCUDA, Ksum.memptr(), count = n * n * sizeof( double ), kind = hipMemcpyHostToDevice);
    hipMemcpy(replicatesCUDA, replicates.memptr(), count = replicates.n_rows * replicates.n_cols * sizeof( double ), 
                kind = hipMemcpyHostToDevice);
    hipMemcpy(sampleCUDA, sample.memptr(), count = n * sizeof( double ), kind = hipMemcpyHostToDevice); 

    // Computing the statistic
    

    // Free resources
    hipFree( hsicCUDA );
    hipFree( replicatesCUDA );
    hipFree( prodCUDA );
    hipFree( sampleCUDA );


    // Transfer data to GPU
    viennacl::matrix<double> hsicCL(n, n);
    viennacl::matrix<double> replicatesCL(replicates.n_rows, replicates.n_cols), prodCL(replicates.n_rows, replicates.n_cols);
    viennacl::vector<double> sampleCL(n);

    copy(Ksum, hsicCL);
    copy(replicates, replicatesCL);
    copy(sample, sampleCL);

    // Compute the statistic
    prodCL = viennacl::linalg::prod(hsicCL, replicatesCL);
    prodCL = viennacl::linalg::element_prod(prodCL, replicatesCL);
    viennacl::vector<double> statCL = viennacl::linalg::column_sum(prodCL);

    copy(statCL, stat);
    double statS = viennacl::linalg::inner_prod(viennacl::linalg::prod(hsicCL, sampleCL), sampleCL);

    double pvalue = arma::sum(stat > statS)/ (double) replicates.n_cols;

    return pvalue;
}
