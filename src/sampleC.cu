#include "hip/hip_runtime.h"
#define ARMA_ALLOW_FAKE_GCC

#define VIENNACL_WITH_CUDA
//#define VIENNACL_WITH_OPENCL
#define VIENNACL_WITH_OPENMP
#define VIENNACL_WITH_ARMADILLO 1

#include <RcppArmadillo.h>
#include <RcppArmadilloExtensions/sample.h>
using namespace Rcpp;


// ViennaCL headers
#include "viennacl/vector.hpp"
#include "viennacl/matrix.hpp"
#include "viennacl/forwards.h"
#include "viennacl/matrix_proxy.hpp"
#include "viennacl/linalg/inner_prod.hpp"


// [[Rcpp::plugins(cpp11)]]
// [[Rcpp::depends(RcppArmadillo)]]

arma::mat sampleC(arma::field<arma::mat> A, NumericVector initial, int n_replicates,
                  double mu = 0.0, double sigma = 1.0,
                  int n_iter = 1.0e+5, int burn_in = 1.0e+3)
{

    int n = initial.size();
    arma::mat qsamples(n, n_replicates + burn_in, arma::fill::zeros);
    arma::mat candidates(n, n_replicates + burn_in + 1, arma::fill::zeros);
    arma::vec candidateO(n), candidateQ(n), candidateN = Rcpp::as<arma::vec>(wrap(pnorm(initial, mu, sigma)));

    // Randomly sample in the sphere unit
    arma::mat theta(n, n_replicates + burn_in, arma::fill::randn);
    theta = arma::normalise(theta, 2, 0);

    // Rejection sampling
    arma::vec cdt(A.n_elem);
    arma::vec::iterator l;
    arma::vec boundA, boundB;
    arma::mat matA(n, n*A.n_elem);
    for (int r = 0; r < A.n_elem; ++r){
        matA(0, n*r, size(A(r))) = A(r); // Regrouping the list of matrices in a single GPU matrix
    }

    // Declaring GPU objects
    viennacl::vector<double> vectorCL(n), resultCL(n);
    viennacl::matrix<double, viennacl::column_major> matrixCL(n, n*A.n_elem);
    copy(matA, matrixCL);

    int r;
    for (int s = 0; s < (n_replicates + burn_in); ++s)
    {
        candidateO = candidateN;

        boundA = -(candidateO/theta.col(s));
        boundB = (1 - candidateO)/theta.col(s);

        double leftQ = std::max(boundA.elem(arma::find(theta.col(s) > 0)).max(),
                                boundB.elem(arma::find(theta.col(s) < 0)).max());
        double rightQ = std::min(boundA.elem(arma::find(theta.col(s) < 0)).min(),
                                 boundB.elem(arma::find(theta.col(s) > 0)).min());

        for (int iter = 0; iter < n_iter; ++iter)
        {
            if (iter == n_iter) stop("The quadratic constraints cannot be satisfied");
            double lambda = runif(1, leftQ, rightQ)[0];
            candidateN = candidateO + lambda * theta.col(s);
            candidateQ = Rcpp::as<arma::vec>(wrap(qnorm(as<NumericVector>(wrap(candidateN)), mu, sigma)));
            viennacl::copy(candidateQ, vectorCL);
            for(l = cdt.begin(), r = 0; l != cdt.end(); ++l, ++r)
            {
                resultCL = viennacl::linalg::prod(viennacl::project(matrixCL, viennacl::range(0, n), viennacl::range(n*r, n*(r+1))), vectorCL);
                *l = viennacl::linalg::inner_prod(vectorCL, resultCL);

            }
            if (all(cdt >= 0)) {
                qsamples.col(s) = candidateQ;
                break;
            }

        }
    }

    return qsamples.cols(burn_in, n_replicates + burn_in - 1);
}
