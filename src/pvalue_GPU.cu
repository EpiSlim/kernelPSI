#include "hip/hip_runtime.h"
#define ARMA_ALLOW_FAKE_GCC

#include <RcppArmadillo.h>
// #include <RcppArmadilloExtensions/sample.h>
using namespace Rcpp;

#include "hsic.h"

// CUDA headers
#include "hipblas.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// CUDA kernels
__global__ void cuda_element_prod(int n, double *x, double *y) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride)
    x[i] *= y[i];
}

__global__ void cuda_column_sum(int n, int p, double *x, double *y) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < p; i += stride) {
    x[i] = 0;
    for (int j = 0; j < n; ++j)
      x[i] += y[i * n + j];
  }
}

// [[Rcpp::plugins(cpp11)]]
// [[Rcpp::depends(RcppArmadillo)]]

double pvalue(arma::vec sample, arma::mat replicates,
              arma::field<arma::mat> K) {

  // Compute the sum kernel
  int n = sample.size();
  arma::vec stat(replicates.n_cols);
  arma::mat Ksum(n, n, arma::fill::zeros);
  for (int r = 0; r < K.n_elem; ++r) {
    Ksum += K(r);
  }
  Ksum = quadHSIC(Ksum);

  // CUDA section
  hipblasHandle_t handle;
  hipblasStatus_t statHandle = hipblasCreate(&handle);

  double *hsicCUDA, *replicatesCUDA, *prodCUDA, *sampleCUDA, *tmpCUDA,
      *statCUDA, *statS;

  // Allocate all our host-side (CPU) and device-side (GPU) data
  hipMallocManaged(&hsicCUDA, n * n * sizeof(double));
  hipMallocManaged(&replicatesCUDA,
                    replicates.n_rows * replicates.n_cols * sizeof(double));
  hipMallocManaged(&sampleCUDA, n * sizeof(double));
  hipMallocManaged(&statCUDA, replicates.n_cols * sizeof(double));
  hipMallocManaged(&statS, sizeof(double));
  hipMalloc(&prodCUDA, replicates.n_rows * replicates.n_cols * sizeof(double));
  hipMalloc(&tmpCUDA, n * sizeof(double));

  // Copy data to CUDA objects
  hipMemcpy(hsicCUDA, Ksum.memptr(), n * n * sizeof(double),
             hipMemcpyHostToDevice);
  hipMemcpy(replicatesCUDA, replicates.memptr(),
             replicates.n_rows * replicates.n_cols * sizeof(double),
             hipMemcpyHostToDevice);
  hipMemcpy(sampleCUDA, sample.memptr(), n * sizeof(double),
             hipMemcpyHostToDevice);

  // Set these constants so we get a simple matrix multiply with hipblasDgemm
  double alpha = 1.0;
  double beta = 0.0;

  // Computing the statistic for replicates
  hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, replicates.n_cols, n, &alpha,
              hsicCUDA, n, replicatesCUDA, replicates.n_rows, &beta, prodCUDA,
              n);

  int blockSize = 256;
  int numBlocks =
      (replicates.n_rows * replicates.n_cols + blockSize - 1) / blockSize;
  cuda_element_prod<<<numBlocks, blockSize>>>(
      replicates.n_rows * replicates.n_cols, prodCUDA, replicatesCUDA);
  cuda_column_sum<<<numBlocks, blockSize>>>(
      replicates.n_rows, replicates.n_cols, statCUDA, prodCUDA);

  // Computing statistic for original sample
  hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, 1, n, &alpha, hsicCUDA, n,
              sampleCUDA, n, &beta, tmpCUDA, n);

  hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 1, 1, n, &alpha, tmpCUDA, 1,
              sampleCUDA, n, &beta, statS, 1);

  hipDeviceSynchronize();

  hipMemcpy(stat.memptr(), statCUDA, replicates.n_cols * sizeof(double),
             hipMemcpyHostToHost);
   Rcout << "First value" << *statS << std::endl;
  // Compute p-value
  double pvalue = arma::sum(stat > *statS) / (double)replicates.n_cols;

  // Free resources
  hipblasDestroy(handle);

  hipFree(hsicCUDA);
  hipFree(replicatesCUDA);
  hipFree(prodCUDA);
  hipFree(sampleCUDA);
  hipFree(tmpCUDA);
  hipFree(statCUDA);
  hipFree(statS);

  return pvalue;
}

